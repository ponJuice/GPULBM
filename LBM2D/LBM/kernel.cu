#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "lbm_kernels.cuh"
#include <stdio.h>
#include <math.h>

#include "lbm_data.h"
#include "lbm_util.h"

void setConstantMemory(){
	LBM::IVector3* h_w;
	h_w = new LBM::IVector3[15];

	LBM::setVector(&h_w[0],  0,  0,  0);
	LBM::setVector(&h_w[1],  1,  0,  0);
	LBM::setVector(&h_w[2], -1,  0,  0);
	LBM::setVector(&h_w[3],  0,  1,  0);
	LBM::setVector(&h_w[4],  0, -1,  0);
	LBM::setVector(&h_w[5],  0,  0,  1);
	LBM::setVector(&h_w[6],  0,  0, -1);
	LBM::setVector(&h_w[7],  1,  1,  1);
	LBM::setVector(&h_w[8], -1, -1, -1);
	LBM::setVector(&h_w[9], -1,  1,  1);
	LBM::setVector(&h_w[10], 1, -1, -1);
	LBM::setVector(&h_w[11], 1, -1,  1);
	LBM::setVector(&h_w[12],-1,  1, -1);
	LBM::setVector(&h_w[13], 1,  1,  1);
	LBM::setVector(&h_w[14],-1, -1, -1);

	hipMemcpyToSymbol(HIP_SYMBOL(w),h_w,sizeof(LBM::IVector3)*15);

	delete[] h_w;
}

void allCudaFree(){
	for(int n = info.offset;n < (info.offset + info.size);n++){
		ce = hipFree(d_point[n].a);
		if(ce != hipSuccess){
			printf("failed hipFree <<point.a>>\n");
			printf("error : %s\n",hipGetErrorString(ce));
		}
	}
	ce = hipFree(d_point);
	if(ce != hipSuccess){
		printf("failed hipFree <<point>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
	}
	ce = hipFree(d_info);
	if(ce != hipSuccess){
		printf("failed hipFree <<info>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
	}
	hipDeviceReset();
}

void safery(hipError_t ce,char str[]){
	if(ce == hipSuccess)
		return;
	printf("%s\n",str);
	printf("error : %s\n",hipGetErrorString(ce));
	allCudaFree();
}

int main(){
	hipError_t ce;

	int device_num = 0;	//TITAN���g�p
	hipDeviceProp_t dev;
	hipGetDeviceProperties(&dev,device_num);

	//�R���X�^���g�������ɌW��w���R�s�[
	setConstantMemory();

	//�v�Z�̈�̏��
	LBM::LatticeInfo info;
	LBM::getLbmLatticeInfo(&info,1,4,4,4,15);
	LBM::printLatticeInfo(&info);

	int threads = Math::min(dev.maxThreadsPerBlock,info.size);//32 * ((info.size - 1) / 32 + 1);
	int block = (int)ceil((double)threads/(double)dev.maxThreadsPerBlock);

	printf("threads : %d\n",threads);
	printf("block : %d\n",block);

	int a_array_size = sizeof(double) * info.direct_num;
	int point_size = sizeof(LBM::LBMPoint);

	int a = sizeof(double*);

	printf("a_size : %d\n",a_array_size);
	printf("point size : %d\n",point_size);

	LBM::LBMPoint* d_point;
	LBM::LBMPoint h_point;
	LBM::LatticeInfo* d_info;

	ce = hipMalloc((void**)&d_point,point_size*info.x_max_out*info.y_max_out*info.z_max_out);
	safery(ce,"failed hipMalloc <<point>>");

	double* _a;
	for(int n = 0;n < info.size;n++){
		//�Q�Ɨ̈���܂߂��C���f�b�N�X���Z�o
		int index_out = info.offset 
			+ n
			+ (n / ( info.x_max_in * info.y_max_in ))
				* (info.x_max_out * info.y_max_out - info.x_max_in * info.y_max_in) 
			+ (n / info.x_max_in) % 2 
				* info.max_speed
				* 2;

		//�e�����̃C���f�b�N�X�Z�o
		int x = index_out % info.y_max_out;
		int y = (index_out / info.x_max_out) % info.y_max_out;
		int z = index_out % (info.x_max_out * info.y_max_out);

		//���z�֐��p�̔z����쐬
		ce = hipMalloc((void**)&_a,a_array_size);
		safery(ce,"failed hipMalloc <<point.a>>");

		//�����l��ݒ�
		h_point.density = 1;
		h_point.vx = 2;
		h_point.vy = 3;
		h_point.vz = 4;
		//�}�X�N�̐ݒ�
		if(x == 0){
			h_point.mask = INFLOW;
		}else if(x == (info.x_max_out - 1)){
			h_point.mask = OUTFLOW;
		}
		if(y == 0){
			h_point.mask = OUTFLOW;
		}else if(y == (info.y_max_out - 1)){
			h_point.mask = OUTFLOW;
		}
		if(z == 0){
			h_point.mask = OUTFLOW;
		}else if(z == (info.z_max_out -1 )){
			h_point.mask = OUTFLOW;
		}

		//���z�֐��p�z����Q�Ƃ���悤�ɂ���
		h_point.a = _a;
		//�����R�s�[
		ce = hipMemcpy(&d_point[index_out],&h_point,point_size,hipMemcpyHostToDevice);
		if(ce != hipSuccess){
			printf("failed hipMemcpy <<point.a>>\n");
			printf("error : %s\n",hipGetErrorString(ce));
			goto allCudaFree;
		}
	}

	int info_size = sizeof(LBM::LatticeInfo);
	ce = hipMalloc((void**)&d_info,info_size);
	if(ce != hipSuccess){
		printf("failed hipMalloc <<info>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
		goto allCudaFree;
	}

	ce = hipMemcpy(d_info,&info,info_size,hipMemcpyHostToDevice);	
	if(ce != hipSuccess){
		printf("failed hipMemcpy <<info>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
		goto allCudaFree;
	}

	//�������̐ݒ�

	//�v�Z���s
	lbm_calc<<<block,threads>>>(d_info,d_point);
	//lbm_test3<<<1,1>>>(d_point,_a,info.direct_num);
	hipDeviceSynchronize();
	ce = hipGetLastError();
	if(ce != hipSuccess){
		printf("failed karnel <<lbm_calc>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
		goto allCudaFree;
	}
	

allCudaFree:
	for(int n = info.offset;n < (info.offset + info.size);n++){
		ce = hipFree(d_point[n].a);
		if(ce != hipSuccess){
			printf("failed hipFree <<point.a>>\n");
			printf("error : %s\n",hipGetErrorString(ce));
		}
	}
	ce = hipFree(d_point);
	if(ce != hipSuccess){
		printf("failed hipFree <<point>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
	}
	ce = hipFree(d_info);
	if(ce != hipSuccess){
		printf("failed hipFree <<info>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
	}
	hipDeviceReset();

	return 0;
}